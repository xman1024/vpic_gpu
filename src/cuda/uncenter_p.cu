#include "hip/hip_runtime.h"
/*
 * Written by:
 *   Iwona Kotlarska, Łukasz Kondraciuk
 *   University of Warsaw
 *   2019 - port to CUDA for SC19 student cluster competition
 *
 */
#include <hip/hip_runtime.h>

#include "uncenter_p.h"
#include "utils.h"

__global__ void uncenter_p_kernel(particle_t* p0,
                                  int n,
                                  interpolator_t* f0,
                                  const float qdt_2mc,
                                  const float qdt_4mc) {
    const float one            = 1.0;
    const float one_third      = 1.0 / 3.0;
    const float two_fifteenths = 2.0 / 15.0;

    float dx, dy, dz, ux, uy, uz;
    float hax, hay, haz, cbx, cby, cbz;
    float v0, v1, v2, v3, v4;
    int ii;

    interpolator_t* f;

    int i            = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    // Process particles for this pipeline.

    for (; i < n; i += stride) {
        particle_t* p = p0 + i;
        dx            = p->dx;  // Load position
        dy            = p->dy;
        dz            = p->dz;
        ii            = p->i;

        f = f0 + ii;  // Interpolate E

        hax = qdt_2mc *
              ((f->ex + dy * f->dexdy) + dz * (f->dexdz + dy * f->d2exdydz));

        hay = qdt_2mc *
              ((f->ey + dz * f->deydz) + dx * (f->deydx + dz * f->d2eydzdx));

        haz = qdt_2mc *
              ((f->ez + dx * f->dezdx) + dy * (f->dezdy + dx * f->d2ezdxdy));

        cbx = f->cbx + dx * f->dcbxdx;  // Interpolate B
        cby = f->cby + dy * f->dcbydy;
        cbz = f->cbz + dz * f->dcbzdz;

        ux = p->ux;  // Load momentum
        uy = p->uy;
        uz = p->uz;

        v0 = qdt_4mc / (float)sqrt(one + (ux * ux + (uy * uy + uz * uz)));
        /**/  // Boris - scalars
        v1 = cbx * cbx + (cby * cby + cbz * cbz);
        v2 = (v0 * v0) * v1;
        v3 = v0 * (one + v2 * (one_third + v2 * two_fifteenths));
        v4 = v3 / (one + v1 * (v3 * v3));
        v4 += v4;

        v0 = ux + v3 * (uy * cbz - uz * cby);  // Boris - uprime
        v1 = uy + v3 * (uz * cbx - ux * cbz);
        v2 = uz + v3 * (ux * cby - uy * cbx);

        ux += v4 * (v1 * cbz - v2 * cby);  // Boris - rotation
        uy += v4 * (v2 * cbx - v0 * cbz);
        uz += v4 * (v0 * cby - v1 * cbx);

        ux += hax;  // Half advance E
        uy += hay;
        uz += haz;

        p->ux = ux;  // Store momentum
        p->uy = uy;
        p->uz = uz;
    }
}

void uncenter_p_pipeline_cuda(particle_t* p,
                              int n,
                              interpolator_t* f0,
                              const float qdt_2mc,
                              const float qdt_4mc) {
    uncenter_p_kernel<<<1024, 1024>>>(p, n, f0, qdt_2mc, qdt_4mc);
}
