#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "center_p.cu"
#include "utils.h"

__global__ void rho_p_kernel(const particle_t* p,
                             const float q_8V,
                             const int np,
                             const int sy,
                             const int sz,
                             field_t* f) {
    float w0, w1, w2, w3, w4, w5, w6, w7, dz;

    int n, v;

    for (n = 0; n < np; n++) {
        // After detailed experiments and studying of assembly dumps, it was
        // determined that if the platform does not support efficient 4-vector
        // SIMD memory gather/scatter operations, the savings from using
        // "trilinear" are slightly outweighed by the overhead of the
        // gather/scatters.

        // Load the particle data

        w0 = p[n].dx;
        w1 = p[n].dy;
        dz = p[n].dz;
        v  = p[n].i;
        w7 = p[n].w * q_8V;

        // Compute the trilinear weights
        // Though the PPE should have hardware fma/fmaf support, it was
        // measured to be more efficient _not_ to use it here.  (Maybe the
        // compiler isn't actually generating the assembly for it.

#define FMA(x, y, z) ((z) + (x) * (y))
#define FNMS(x, y, z) ((z) - (x) * (y))
        w6 = FNMS(w0, w7, w7);  // q(1-dx)
        w7 = FMA(w0, w7, w7);   // q(1+dx)
        w4 = FNMS(w1, w6, w6);
        w5 = FNMS(w1, w7, w7);  // q(1-dx)(1-dy), q(1+dx)(1-dy)
        w6 = FMA(w1, w6, w6);
        w7 = FMA(w1, w7, w7);  // q(1-dx)(1+dy), q(1+dx)(1+dy)
        w0 = FNMS(dz, w4, w4);
        w1 = FNMS(dz, w5, w5);
        w2 = FNMS(dz, w6, w6);
        w3 = FNMS(dz, w7, w7);
        w4 = FMA(dz, w4, w4);
        w5 = FMA(dz, w5, w5);
        w6 = FMA(dz, w6, w6);
        w7 = FMA(dz, w7, w7);
#undef FNMS
#undef FMA

        // Reduce the particle charge to rhof

        f[v].rhof += w0;
        f[v + 1].rhof += w1;
        f[v + sy].rhof += w2;
        f[v + sy + 1].rhof += w3;
        f[v + sz].rhof += w4;
        f[v + sz + 1].rhof += w5;
        f[v + sz + sy].rhof += w6;
        f[v + sz + sy + 1].rhof += w7;
    }
}

void rho_p_cuda(const particle_t* p,
                const float q_8V,
                const int np,
                const int sy,
                const int sz,
                field_t* f) {
    rho_p_kernel<<<1, 1>>>(p, q_8V, np, sy, sz, f);
}
